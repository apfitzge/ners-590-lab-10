#include "hip/hip_runtime.h"
#include <chrono>               // Timing
#include <stdio.h>              // printf
#include <algorithm>            // Max (result checking)
#include "lab10_cuda_common.h"  // Error checking macro

static constexpr const int BLOCK_SIZE = 16;

/**
 * \brief Perform matrix multiplicat C = A * B.
 * \param[in] C The resulting matrix (N by P)
 * \param[in] A The left matrix (N by M)
 * \param[in] B The right matrix (M by P)
 * \param[in] N Number of rows in A and C.
 * \param[in] M Number of columns in A and rows in B.
 * \param[in] P Number of columns in B and C.
 */
__global__ void matmul(double C[], const double A[], const double B[],
                       const int N, const int M, const int P) {
    // Compute row and column in global matrix
    const auto row = blockIdx.y * blockDim.y + threadIdx.y;
    const auto col = blockIdx.x * blockDim.x + threadIdx.x;

    // If the element is in the matrix
    if(row < N && col < P) {
        // Local variable to store element in C at (row,col)
        double elem = 0.0;
        // Loop over the columns in A (rows of B)
        for(int idx = 0, idx_A = row * N, idx_B = col; idx < M;
                ++idx, ++idx_A, idx_B += N)
            elem += A[idx_A] * B[idx_B];

        // Store the result in C
        C[row * N + col] = elem;
    }
}

/** Main routine */
int main(int argc, char** argv) {
    // Read input
    if(argc < 4) {
        printf("Must enter matrix dimensions: N, M, P!\n");
        exit(1);
    }
    const int N = atoi(argv[1]);
    const int M = atoi(argv[2]);
    const int P = atoi(argv[3]);
    const int repeat = (argc >= 5) ? atoi(argv[4]) : 1;

    // Get matrix sizes
    const auto A_size = N * M * sizeof(double);
    const auto B_size = M * P * sizeof(double);
    const auto C_size = N * P * sizeof(double);

    // Allocate host-side memory
    double* const h_A = (double*) malloc(A_size);
    double* const h_B = (double*) malloc(B_size);
    double* const h_C = (double*) malloc(C_size);

    // Allocate device-side memory
    double *d_A, *d_B, *d_C;
    gpuErrCheck( hipMalloc(&d_A, A_size) );
    gpuErrCheck( hipMalloc(&d_B, B_size) );
    gpuErrCheck( hipMalloc(&d_C, C_size) );

    // Initialize host-side memory
    // A - each row has row index (as double)
    for(int row = 0, idx = 0; row < N; ++row)
        for(int col = 0; col < M; ++col, ++idx)
            h_A[idx] = static_cast<double>(row);
    // B - each col has col index (as double)
    for(int row = 0, idx = 0; row < N; ++row)
        for(int col = 0; col < P; ++col, ++idx)
            h_B[idx] = static_cast<double>(col);

    auto t1 = std::chrono::high_resolution_clock::now();

    // Transfer the matrices to the device
    gpuErrCheck( hipMemcpy(d_A, h_A, A_size, hipMemcpyHostToDevice) );
    gpuErrCheck( hipMemcpy(d_B, h_B, B_size, hipMemcpyHostToDevice) );

    const auto num_blocks_x = (P + BLOCK_SIZE - 1) / BLOCK_SIZE;
    const auto num_blocks_y = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    const dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    const dim3 grid(num_blocks_x, num_blocks_y);

    for(int iter = 0; iter < repeat; ++iter) {
        // Call matmul kernel
        matmul<<<grid, block>>>(d_C, d_A, d_B, N, M, P);

        // Wait for calculation to finish
        gpuErrCheck( hipPeekAtLastError() );
        gpuErrCheck( hipDeviceSynchronize() );
    }

    // Transfer the result matrix from device to host
    gpuErrCheck( hipMemcpy(h_C, d_C, C_size, hipMemcpyDeviceToHost) );

    // End timer
    auto t2 = std::chrono::high_resolution_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::duration<double>>(t2-t1).count();

    // Check the result
    double maxError = 0.0;
    double db_A_cols = static_cast<double>(M);
    for(int row = 0, idx = 0; row < N; ++row) {
      for(int col = 0; col < P; ++col, ++idx) {
        double expected = db_A_cols * row * col;
        maxError = std::max(maxError, std::abs(expected - h_C[idx]));
      }
    }
    if(maxError > 1.0e-8){
      printf(" Result does not match!\n");
      exit(1);
    }

    // Compute FLOPs
    double FLOPs = 2 * double(N) * double(M) * double(P) * double(repeat);
    double GFLOPS = 1.0e-9 * FLOPs / time;

    printf("Problem:\n");
    printf("  Dimensions - N(%d) M(%d) P(%d) repeated %d times\n", N, M, P, repeat);
    printf("  operations=( %g ) time=( %g s ) GFLOPs=( %g )\n", FLOPs, time, GFLOPS);

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C);
    gpuErrCheck( hipFree(d_A) );
    gpuErrCheck( hipFree(d_B) );
    gpuErrCheck( hipFree(d_C) );
}