
#include <hip/hip_runtime.h>
#include <chrono>       // Timing
#include <stdio.h>      // printf
#include <cmath>        // AtomicAdd (hint)

// Block size
static constexpr const int BLOCK_SIZE = 256;

// Error checking in CUDA for allocation and data passing
// Copied from: https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrCheck(ans) {gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
        bool abort=true) {
    if(code != hipSuccess) {
        fprintf(stderr, "GPUAssert: %s %s %d\n", hipGetErrorString(code), file, line);
        if(abort) exit(code);
    }
}
/**
 * \brief Cuda kernel to compute the dot-product of two vectors of length N
 * \param[out] ans The result
 * \param[in] A The first vector
 * \param[in] B The second vector
 * \param[in] N The vector length
 */
__global__ void dot(double* ans, const double A[], const double B[], const int N) {
    // Implement a dot-product routine
}

/** Main routine */
int main(int argc, char** argv) {
    // Read input
    int N = 0;
    if(argc >= 2) {
        N = atoi(argv[1]);
    } else {
        printf("Enter the vector length N.\n");
    }

    // Set up calculation
    const int vs = N * sizeof(double);

    // Allocate memory on the host (CPU)
    double* const h_A = (double*) malloc(vs);
    double* const h_B = (double*) malloc(vs);
    double* const h_p = (double*) malloc(sizeof(double));
    *h_p = 0.0;

    // Allocate memory on the device (GPU)
    double *d_A, *d_B, *d_p;
    gpuErrCheck( hipMalloc(&d_A, vs) );
    gpuErrCheck( hipMalloc(&d_B, vs) );
    gpuErrCheck( hipMalloc(&d_p, vs) );

    // Initialize A and B on the host
    for(int idx = 0; idx < N; ++idx) {
        h_A[idx] = 2.0;
        h_B[idx] = 3.0;
    }
    // Begin timer
    auto t1 = std::chrono::high_resolution_clock::now();

    // Call the dot-product kernel
    //      Hint: You should think about where the memory of each variable lives

    // End timer
    auto t2 = std::chrono::high_resolution_clock::now();
    auto time = std::chrono::duration_cast<std::chrono::duration<double>>(t2-t1).count();

    // Compute FLOPs
    double FLOPs = 2 * static_cast<double>(N);
    double GFLOPS = 1.0e-9 * FLOPs / time;
    printf("Problem:\n");
    printf("  Result: %lf\n", *h_p);
    printf("  Dimensions - N(%d)\n", N);
    printf("  operations=( %g ) time=( %g s ) GFLOPs=( %g )\n", FLOPs, time, GFLOPS);

    // Free memory
    free(h_A);
    free(h_B);
    gpuErrCheck( hipFree(d_A) );
    gpuErrCheck( hipFree(d_B) );
}